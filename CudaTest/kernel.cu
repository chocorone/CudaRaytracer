#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"
#include ""

#include <cmath>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "src/core/hitable.h"
#include "src/core/camera.h"
#include "src/mesh/obj_loader.h"

#define RESOLUTION 1
#define SAMPLES 100

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)


void check_cuda(hipError_t result,
    char const* const func,
    const char* const file,
    int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__global__ void random_init(int nx,
    int ny,
    hiprandState* state) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= nx) || (y >= ny)) return;
    int pixel_index = y * nx + x;
    hiprand_init(0, pixel_index, 0, &state[pixel_index]);
}

__global__ void destroy(Hitable** obj_list,
    Hitable** world,
    Camera** camera,
    int obj_cnt) {
    for (int i = 0; i < obj_cnt; i++) {
        delete* (obj_list + i);
    }
    delete* world;
    delete* camera;
}

__global__ void renderTest(vec3* colorBuffer,
    hiprandState* state,
    int nx,
    int ny) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= nx) || (y >= ny)) return;

    int pixel_index = y * nx + x;
    vec3 col(0, 0, 0);

    col[0] = float(x) / float(nx);
    col[1] = float(y) / float(ny);
    col[2] = 0.5f;

    colorBuffer[pixel_index] = clip(col);
}


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    std::ofstream imgWrite("images/image.ppm");

    int nx = 1024 * RESOLUTION;
    int ny = 512 * RESOLUTION;
    int tx = 16;
    int ty = 16;

    int num_pixel = nx * ny;


    // 画素のメモリ確保
    vec3* colorBuffer;

    checkCudaErrors(hipMallocManaged((void**)&colorBuffer, num_pixel * sizeof(vec3)));

    // 乱数列生成用のメモリ確保
    hiprandState* curand_state;
    checkCudaErrors(hipMallocManaged((void**)&curand_state, num_pixel * sizeof(hiprandState)));

    // シーン作成
    int obj_cnt = 488;
    Hitable** obj_list;
    Hitable** world;
    Camera** camera;
    checkCudaErrors(hipMallocManaged((void**)&obj_list, obj_cnt * sizeof(Hitable*)));
    checkCudaErrors(hipMallocManaged((void**)&world, sizeof(Hitable*)));
    checkCudaErrors(hipMallocManaged((void**)&camera, sizeof(Camera*)));

    
    // 画素ごとに乱数を初期化
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    random_init << <blocks, threads >> > (nx, ny, curand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    
    // --------------------------- allocate the mesh ----------------------------------------
    vec3* points;
    vec3* idxVertex;

    // NOTE: must pre-allocate before initialize the elements
    checkCudaErrors(hipMallocManaged((void**)&points, 2600 * sizeof(vec3)));
    checkCudaErrors(hipMallocManaged((void**)&idxVertex, 5000 * sizeof(vec3)));

    int nPoints, nTriangles;
    parseObjByName("./objects/small_bunny.obj", points, idxVertex, nPoints, nTriangles);

    std::cout << "# of points: " << nPoints << std::endl;
    std::cout << "# of triangles: " << nTriangles << std::endl;

    // 大きくしてる？
    for (int i = 0; i < nPoints; i++) { points[i] *= 30.0; }
    for (int i = 0; i < nPoints; i++) { std::cout << points[i] << std::endl; }

    /*
    Hitable** triangles;
    checkCudaErrors(hipMallocManaged((void**)&triangles, nTriangles * sizeof(Hitable*)));
    // --------------------------- ! allocate the mesh ---------------------------------------

    // オブジェクト、カメラの生成
    build_mesh << <1, 1 >> > (world, camera, triangles, points,
        idxVertex, nPoints, nTriangles, curand_state, nx, ny, obj_cnt);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    */

    // レンダリング
    //render << <blocks, threads >> > (colorBuffer, world, camera, curand_state, nx, ny, SAMPLES);
    renderTest <<<blocks,threads>>> (colorBuffer,curand_state, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // 画像書き出し
    imgWrite << "P3\n" << nx << " " << ny << "\n255\n";
    for (int i = ny - 1; i >= 0; i--) {
        for (int j = 0; j < nx; j++) {
            size_t pixel_index = i * nx + j;
            int ir = int(255.99 * colorBuffer[pixel_index].r());
            int ig = int(255.99 * colorBuffer[pixel_index].g());
            int ib = int(255.99 * colorBuffer[pixel_index].b());
            imgWrite << ir << " " << ig << " " << ib << "\n";
        }
    }
    
    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    destroy << <1, 1 >> > (obj_list, world, camera, obj_cnt);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(obj_list));
    checkCudaErrors(hipFree(camera));
    checkCudaErrors(hipFree(curand_state));
    checkCudaErrors(hipFree(colorBuffer));
    

    hipDeviceReset();

    


    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
