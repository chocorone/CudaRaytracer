﻿#include "core/render.h"


int main()
{
    // パラメーター設定
    const int nx = 1024 * RESOLUTION;
    const int ny = 512 * RESOLUTION;  
    const int threadX = 16;
    const int threadY = 16;
    const int max_depth = 8;
    const int samples = 4;
    const int beginFrame = 0;
    const int endFrame = 0;

    const int num_pixel = nx * ny;

    //ヒープサイズ・スタックサイズ指定
    ChangeHeapSize(1024 * 1024 * 1024);
    ChangeStackSize(4096 * 2);
    // 乱数列生成用のメモリ確保
    hiprandState* curand_state;
    checkCudaErrors(hipMallocManaged((void**)&curand_state, num_pixel * sizeof(hiprandState)));

    //シーン保存用の変数のメモリ確保
    HitableList** world;
    Camera** camera;
    TransformList** transformPointer;
    checkCudaErrors(hipMallocManaged((void**)&world, sizeof(HitableList*)));
    checkCudaErrors(hipMallocManaged((void**)&camera, sizeof(Camera*)));
    checkCudaErrors(hipMallocManaged((void**)&transformPointer, sizeof(TransformList*)));
    BVHNode** bvh;
    checkCudaErrors(hipMallocManaged((void**)&bvh, sizeof(BVHNode*)));
    
    // 画素ごとに乱数を初期化
    dim3 blocks(nx / threadX + 1, ny / threadY + 1);
    dim3 threads(threadX, threadY);
    random_init <<<blocks, threads >>> (nx, ny, curand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    //FBXファイル読み込み
    MeshData* meshData;
    checkCudaErrors(hipMallocManaged((void**)&meshData, sizeof(MeshData*)));
    CreateFBXMeshData("./objects/HipHopDancing.fbx", meshData);
    CreateFBXMeshData("./objects/bunny2.fbx", meshData);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // オブジェクト、カメラの生成
    AnimationDataList* animationData = new AnimationDataList();
    create_camera << <1, 1 >> > (camera, nx, ny, vec3(0,0,20), vec3(0, 0, 0), 10.0, 0.0, 60);
    //create_camera << <1, 1 >> > (camera, nx, ny, vec3(278, 278, -700), vec3(278, 278, 0), 10.0, 0.0, 40);
    init_data << <1, 1 >> > (world, transformPointer);
    //BuildAnimatedSphere(world,animationData, transformPointer);
    add_mesh_withNormal << <1, 1 >> > (world, meshData, transformPointer);
    CHECK(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    printf("シーン作成完了\n");

    //BVHの作成
    create_BVH << <1, 1 >> > (world, bvh, curand_state);
    CHECK(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    printf("BVH作成完了\n");

    //レンダリング
    //renderAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)world, camera, animationData, transformPointer, blocks, threads, curand_state);
    renderAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)bvh, camera,animationData,transformPointer,blocks,threads,curand_state);
    
    //メモリ解放
    animationData->freeMemory();
    checkCudaErrors(hipDeviceSynchronize());
    destroy << <1, 1 >> > (world, camera, transformPointer);
    destroy << <1, 1 >> > (meshData);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(camera));
    checkCudaErrors(hipFree(transformPointer));
    checkCudaErrors(hipFree(curand_state));
    checkCudaErrors(hipFree(meshData));
    checkCudaErrors(hipFree(bvh));

    
    hipDeviceReset();


    return 0;
}

