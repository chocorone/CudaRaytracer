﻿#include "core/render.h"


class HostPointerList
{
public:
    HostPointerList() { list = new void* (); list_size = 0; }
    HostPointerList(void** l, int n) { list = l; list_size = n; }
    void append(void** data)
    {
        void** tmp = (void**)malloc(sizeof(void*) * list_size);

        for (int i = 0; i < list_size; i++)
        {
            tmp[i] = list[i];
        }

        free(list);

        list_size++;

        list = (void**)malloc(sizeof(void*) * list_size);

        for (int i = 0; i < list_size - 1; i++)
        {
            list[i] = tmp[i];
        }
        list[list_size - 1] = data;

        free(tmp);
    }
    void freeMemory()
    {
        for (size_t i = 0; i < list_size; i++)
        {
            free(list[i]);
        }
        free(list);
        list_size = 0;
    }
    void** list;
    int list_size;
};

int main()
{
    // パラメーター設定
    const int nx = 1024 * RESOLUTION;
    const int ny = 512 * RESOLUTION;  
    const int threadX = 16;
    const int threadY = 16;
    const int max_depth = 8;
    const int samples = 4;
    const int beginFrame = 0;
    const int endFrame = 90;

    const int num_pixel = nx * ny;
    dim3 blocks(nx / threadX + 1, ny / threadY + 1);
    dim3 threads(threadX, threadY);
    CudaPointerList* pointerList = new CudaPointerList();//あとで破棄するデバイス用ポインターのリスト

    //ヒープサイズ・スタックサイズ指定
    ChangeHeapSize(1024 * 1024 * 1024);
    ChangeStackSize(4096 * 2);
    // 乱数列生成用のメモリ確保
    hiprandState* curand_state;
    checkCudaErrors(hipMallocManaged((void**)&curand_state, nx * ny * sizeof(hiprandState)));
    SetCurandState(curand_state, nx, ny, blocks, threads);
    pointerList->append((void**)curand_state);

    //カメラ作成
    Camera** camera;
    checkCudaErrors(hipMallocManaged((void**)&camera, sizeof(Camera*)));
    pointerList->append((void**)camera);
    create_camera << <1, 1 >> > (camera, nx, ny, vec3(0, 20, 400), vec3(0, 20, 0), 10.0, 0.0, 60);
    //create_camera << <1, 1 >> > (camera, nx, ny, vec3(278, 278, -700), vec3(278, 278, 0), 10.0, 0.0, 40);

    //あとで修正
    HitableList** world;
    TransformList** transformPointer;
    checkCudaErrors(hipMallocManaged((void**)&world, sizeof(HitableList*)));
    checkCudaErrors(hipMallocManaged((void**)&transformPointer, sizeof(TransformList*)));
    init_data << <1, 1 >> > (world, transformPointer);
    AnimationDataList* animationData = new AnimationDataList();
    //BuildAnimatedSphere(world,animationData, transformPointer);
    CHECK(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    
    //FBXオブジェクト作成
    FBXObject* fbxData = new FBXObject();//モデルデータ
    checkCudaErrors(hipMallocManaged((void**)&fbxData, sizeof(FBXObject*)));
    pointerList->append((void**)fbxData);
    FBXAnimationData* fbxAnimationData;//アニメーションデータ
    fbxAnimationData = (FBXAnimationData*)malloc(sizeof(FBXAnimationData*));
    CreateFBXData("./objects/human_light.fbx", fbxData, fbxAnimationData);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    add_mesh_fromPoseData << <1, 1 >> > (world, fbxData, fbxAnimationData->animation[0]); //メッシュの移動と作成
    BVHNode** bvh;
    checkCudaErrors(hipMallocManaged((void**)&bvh, sizeof(BVHNode*)));
    pointerList->append((void**)bvh);
    create_BVH << <1, 1 >> > (world, bvh, curand_state);
    CHECK(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    printf("BVH作成完了\n");

    //レンダリング
    //renderAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)world, camera, animationData, transformPointer, fbxAnimationData, blocks, threads, curand_state);
    renderAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)bvh, camera,animationData,transformPointer, fbxAnimationData,blocks,threads,curand_state);
    
    //メモリ解放
    checkCudaErrors(hipDeviceSynchronize());
    //destroy << <1, 1 >> > (world, camera, transformPointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(transformPointer));
    pointerList->freeMemory();
    free(pointerList);
    free(animationData);
    free(fbxAnimationData);

    
    hipDeviceReset();


    return 0;
}

