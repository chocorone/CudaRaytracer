﻿#include "Loader/CSVWriter.h"
#include "core/render.h"


void renderBoneBVH(int nx, int ny, int samples, int max_depth, int beginFrame, int endFrame,
    Camera** camera, 
    dim3 blocks, dim3 threads, hiprandState* curand_state, std::vector<std::vector<std::string>>& data,
    CudaPointerList* pointerList, FBXObject* fbxData)
{
    StopWatch sw;
    sw.Reset();
    sw.Start();
    HitableList** boneBVHList;
    hipMalloc(&boneBVHList, sizeof(HitableList*));
    createBoneBVH(boneBVHList, fbxData, curand_state, pointerList);
    sw.Stop();
    printf("BVH作成完了\n");
    data.push_back({ "", "", "",std::to_string(sw.GetTime()) });
    renderBVHNodeAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)boneBVHList, camera, fbxData, blocks, threads, curand_state, data);

}


void renderBVH(int nx, int ny, int samples, int max_depth, int beginFrame, int endFrame,
    HitableList** fbxList, Camera** camera,
    dim3 blocks, dim3 threads, hiprandState* curand_state, std::vector<std::vector<std::string>>& data,
    CudaPointerList* pointerList, FBXObject* fbxData)
{
    StopWatch sw;
    sw.Reset();
    sw.Start();
    BVHNode** bvhNode;
    hipMalloc(&bvhNode, sizeof(BVHNode*));
    create_BVHfromList(bvhNode, fbxList, curand_state, pointerList);
    sw.Stop();
    printf("BVH作成完了\n");
    data.push_back({ "", "", "",std::to_string(sw.GetTime()) });
    renderBVHAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)bvhNode, camera, fbxData, blocks, threads, curand_state, data);
}

int main()
{
    // パラメーター設定
    const int nx = 1024 * RESOLUTION;
    const int ny = 512 * RESOLUTION;  
    const int threadX = 16;
    const int threadY = 16;
    const int max_depth = 8;
    const int samples = 4;
    const int beginFrame = 0;
    int endFrame = 30;

    const int num_pixel = nx * ny;
    dim3 blocks(nx / threadX + 1, ny / threadY + 1);
    dim3 threads(threadX, threadY);
    CudaPointerList* pointerList = new CudaPointerList();//あとで破棄するデバイス用ポインターのリスト

    //計測用データ
    StopWatch sw;
    std::vector<std::vector<std::string>> data;
    data.push_back({ "frame", "rendering", "update","build"});

    //ヒープサイズ・スタックサイズ指定
    //ChangeHeapSize(1024 * 1024 * 1024*4);
    hipError_t err = hipDeviceSetLimit(hipLimitMallocHeapSize, 1048576ULL * 2048);

    ChangeStackSize(1024 * 16);
    // 乱数列生成用のメモリ確保
    hiprandState* curand_state;
    checkCudaErrors(hipMallocManaged((void**)&curand_state, nx * ny * sizeof(hiprandState)));
    SetCurandState(curand_state, nx, ny, blocks, threads,pointerList);

    //カメラ作成
    Camera** camera;
    checkCudaErrors(hipMallocManaged((void**)&camera, sizeof(Camera*)));
    init_camera(camera, nx, ny, pointerList);

    //オブジェクト作成
    TransformList** transformPointer;
    checkCudaErrors(hipMallocManaged((void**)&transformPointer, sizeof(TransformList*)));
    init_TransformList(transformPointer, pointerList);
    AnimationDataList* animationData = new AnimationDataList();

    //FBXオブジェクト作成
    HitableList** fbxList;
    checkCudaErrors(hipMallocManaged((void**)&fbxList, sizeof(HitableList*)));
    init_List(fbxList, pointerList);
    //FBXファイル読み込み
    FBXObject* fbxData = new FBXObject();//モデルデータ
    //create_FBXObject("./objects/low_walking.fbx", fbxData, fbxAnimationData, endFrame, pointerList);
    //CreateFBXData("./objects/high_Walking5.fbx", fbxData, endFrame);
    CreateFBXData("./objects/high_StandUp2.fbx", fbxData, endFrame);
    // メッシュの生成
    create_FBXMesh(fbxList, fbxData);
    printf("シーン作成完了\n");

    //endFrame = 0;
    //ただのリスト
    //renderListAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)fbxList, camera, fbxAnimationData, blocks, threads, curand_state);
    //ボーンによるBVH
    //renderBoneBVH(nx, ny, samples, max_depth, beginFrame, endFrame, camera, blocks, threads, curand_state, data, pointerList, fbxData);
    //BVH
    renderBVH(nx, ny, samples, max_depth, beginFrame, endFrame, fbxList, camera, blocks, threads, curand_state, data, pointerList,fbxData);


    // CSVファイルに書き出す
    writeCSV("output.csv", data);
    printf("csv書き出し完了\n");

    //メモリ解放
    checkCudaErrors(hipDeviceSynchronize());
    pointerList->freeMemory();
    hipDeviceReset();
    checkCudaErrors(hipGetLastError());

    free(animationData);

    return 0;
}

