﻿#include "Loader/CSVWriter.h"
#include "core/render.h"


void renderBoneBVH(int nx, int ny, int samples, int max_depth, int beginFrame, int endFrame,
    Camera** camera, FBXAnimationData* fbxAnimationData,
    dim3 blocks, dim3 threads, hiprandState* curand_state, std::vector<std::vector<std::string>>& data,
    CudaPointerList* pointerList, FBXObject* fbxData)
{
    StopWatch sw;
    sw.Reset();
    sw.Start();
    HitableList** boneBVHList;
    checkCudaErrors(hipMallocManaged((void**)&boneBVHList, sizeof(HitableList**)));
    //init_List(boneBVHList, pointerList);
    createBoneBVH(boneBVHList, fbxData, curand_state, pointerList);
    sw.Stop();
    printf("BVH作成完了\n");
    data.push_back({ "", "", "",std::to_string(sw.GetTime()) });
    renderBVHNodeAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)boneBVHList, camera, fbxAnimationData, blocks, threads, curand_state, data);

}


void renderBVH(int nx, int ny, int samples, int max_depth, int beginFrame, int endFrame,
    HitableList** fbxList, Camera** camera, FBXAnimationData* fbxAnimationData,
    dim3 blocks, dim3 threads, hiprandState* curand_state, std::vector<std::vector<std::string>>& data,
    CudaPointerList* pointerList)
{
    StopWatch sw;
    sw.Reset();
    sw.Start();
    BVHNode** bvhNode;
    checkCudaErrors(hipMallocManaged((void**)&bvhNode, sizeof(HitableList**)));
    create_BVHfromList(bvhNode, fbxList, curand_state, pointerList);
    sw.Stop();
    printf("BVH作成完了\n");
    data.push_back({ "", "", "",std::to_string(sw.GetTime()) });
    renderBVHAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)bvhNode, camera, fbxAnimationData, blocks, threads, curand_state, data);
}

int main()
{
    // パラメーター設定
    const int nx = 1024 * RESOLUTION;
    const int ny = 512 * RESOLUTION;  
    const int threadX = 16;
    const int threadY = 16;
    const int max_depth = 8;
    const int samples = 4;
    const int beginFrame = 0;
    int endFrame = 30;

    const int num_pixel = nx * ny;
    dim3 blocks(nx / threadX + 1, ny / threadY + 1);
    dim3 threads(threadX, threadY);
    CudaPointerList* pointerList = new CudaPointerList();//あとで破棄するデバイス用ポインターのリスト

    //計測用データ
    std::vector<std::vector<std::string>> data;
    data.push_back({ "frame", "rendering", "update","build"});

    //ヒープサイズ・スタックサイズ指定
    ChangeHeapSize(1024 * 1024 * 1024 * 128);
    ChangeStackSize(1024 * 128);
    // 乱数列生成用のメモリ確保
    hiprandState* d_curand_state;
    hipMalloc(&d_curand_state, nx * ny * sizeof(hiprandState));
    SetCurandState(d_curand_state, nx, ny, blocks, threads,pointerList);

    //カメラ作成
    Camera** d_camera;
    hipMalloc(&d_camera, sizeof(Camera*));
    init_camera(d_camera, nx, ny, pointerList);

    //FBXオブジェクト作成
    HitableList** fbxList;
    checkCudaErrors(hipMallocManaged((void**)&fbxList, sizeof(HitableList*)));
    init_List(fbxList, pointerList);
    //FBXファイル読み込み
    FBXObject* fbxData = new FBXObject();//モデルデータ
    checkCudaErrors(hipMallocManaged((void**)&fbxData, sizeof(FBXObject*)));
    FBXAnimationData* fbxAnimationData;//アニメーションデータ
    fbxAnimationData = new FBXAnimationData();
    //create_FBXObject("./objects/high_Walking3.fbx", fbxData, fbxAnimationData, endFrame, pointerList);
    create_FBXObject("./objects/low_walking.fbx", fbxData, fbxAnimationData, endFrame, pointerList);
    //create_FBXObject("./objects/low_standUp.fbx", fbxData, fbxAnimationData, endFrame, pointerList);
    
    // メッシュの生成
    create_FBXMesh(fbxList, fbxData, fbxAnimationData);
    printf("シーン作成完了\n");

    //ただのリスト
    //renderListAnimation(nx, ny, samples, max_depth, beginFrame, endFrame, (Hitable**)fbxList, camera, fbxAnimationData, blocks, threads, curand_state);
    //ボーンによるBVH
    //renderBoneBVH(nx, ny, samples, max_depth, beginFrame, endFrame, camera, fbxAnimationData, blocks, threads, curand_state, data, pointerList, fbxData);
    //BVH
    renderBVH(nx, ny, samples, max_depth, beginFrame, endFrame, fbxList, d_camera, fbxAnimationData, blocks, threads, d_curand_state, data, pointerList);


    // CSVファイルに書き出す
    writeCSV("output.csv", data);
    printf("csv書き出し完了\n");

    //メモリ解放
    checkCudaErrors(hipDeviceSynchronize());
    pointerList->freeMemory();
    hipDeviceReset();
    checkCudaErrors(hipGetLastError());
    free(fbxAnimationData);

    return 0;
}

